/* Copyright 2025 Grup Mediapro S.L.U (Oscar Amoros Huguet)

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <tests/main.h>

#include <benchmarks/fkBenchmarksCommon.h>
#include <benchmarks/twoExecutionsBenchmark.h>

#include <fused_kernel/fused_kernel.cuh>

constexpr char VARIABLE_DIMENSION_NAME[]{ "Batch size" };

constexpr size_t NUM_EXPERIMENTS = 60; // Used 100 in the paper
constexpr size_t FIRST_VALUE = 1;
constexpr size_t INCREMENT = 5;

constexpr std::array<size_t, NUM_EXPERIMENTS> variableDimensionValues = arrayIndexSecuence<FIRST_VALUE, INCREMENT, NUM_EXPERIMENTS>;

template <int BATCH>
bool benchark_Horizontal_Fusion(const size_t& NUM_ELEMS_X, const size_t& NUM_ELEMS_Y, const hipStream_t& stream) {
    std::stringstream error_s;
    bool passed = true;
    bool exception = false;

    using InputType = uchar;
    using OutputType = float;

    const uchar val_init = 10u;
    const float val_alpha = 1.0f;
    const float val_sub = 1.f;
    const float val_div = 3.2f;
    try {
        const fk::Size cropSize(60, 120);
        fk::Ptr2D<InputType> d_input((int)NUM_ELEMS_Y, (int)NUM_ELEMS_X);
        fk::setTo(val_init, d_input, stream);
        std::array<fk::Ptr2D<float>, BATCH> d_output_cv;
        std::array<fk::Ptr2D<float>, BATCH> h_cvResults;
        std::array<fk::Ptr2D<float>, BATCH> h_cvGSResults;

        cv::cuda::GpuMat d_temp(cropSize, CV_TYPE_O);
        cv::cuda::GpuMat d_temp2(cropSize, CV_TYPE_O);

        cv::cuda::GpuMat d_tensor_output(BATCH,
            cropSize.width * cropSize.height,
            CV_TYPE_O);
        d_tensor_output.step = cropSize.width * cropSize.height * sizeof(CUDA_T(CV_TYPE_O));

        cv::Mat diff(cropSize, CV_TYPE_O);
        cv::Mat h_tensor_output(BATCH, cropSize.width * cropSize.height, CV_TYPE_I);

        std::array<cv::cuda::GpuMat, BATCH> crops;
        for (int crop_i = 0; crop_i < BATCH; crop_i++) {
            crops[crop_i] = cv::cuda::GpuMat(cropSize, CV_TYPE_I, val_init);
            d_output_cv[crop_i].create(cropSize, CV_TYPE_O);
            h_cvResults[crop_i].create(cropSize, CV_TYPE_O);
        }

        START_FIRST_BENCHMARK
            for (int crop_i = 0; crop_i < BATCH; crop_i++) {
                cvGS::executeOperations<false>(crops[crop_i], cv_stream,
                    cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>((float)alpha),
                    cvGS::subtract<CV_TYPE_O>(val_sub),
                    cvGS::divide<CV_TYPE_O>(val_div),
                    cvGS::write<CV_TYPE_O>(d_output_cv[crop_i]));
            }

        STOP_FIRST_START_SECOND_BENCHMARK
            // cvGPUSpeedup
            // Assuming we use all the batch
            // On Linux it is necessary to pass the BATCH as a template parameter
            // On Windows (VS2022 Community) it is not needed, it is deduced from crops 
            cvGS::executeOperations<false, BATCH>(crops, cv_stream,
                cvGS::convertTo<CV_TYPE_I, CV_TYPE_O>((float)alpha),
                cvGS::subtract<CV_TYPE_O>(val_sub),
                cvGS::divide<CV_TYPE_O>(val_div),
                cvGS::write<CV_TYPE_O>(d_tensor_output, cropSize));

        STOP_SECOND_BENCHMARK

            d_tensor_output.download(h_tensor_output, cv_stream);

        // Verify results
        for (int crop_i = 0; crop_i < BATCH; crop_i++) {
            d_output_cv[crop_i].download(h_cvResults[crop_i], cv_stream);
        }

        cv_stream.waitForCompletion();

        for (int crop_i = 0; crop_i < BATCH; crop_i++) {
            cv::Mat cvRes = h_cvResults[crop_i];
            cv::Mat cvGSRes = cv::Mat(cropSize.height, cropSize.width, CV_TYPE_O, h_tensor_output.row(crop_i).data);
            bool passedThisTime = compareAndCheck<CV_TYPE_O>(cropSize.width, cropSize.height, cvRes, cvGSRes);
            if (!passedThisTime) { std::cout << "Failed on crop idx=" << crop_i << std::endl; }
            passed &= passedThisTime;
        }
    } catch (const cv::Exception& e) {
        if (e.code != -210) {
            error_s << e.what();
            passed = false;
            exception = true;
        }
    } catch (const std::exception& e) {
        error_s << e.what();
        passed = false;
        exception = true;
    }

    if (!passed) {
        if (!exception) {
            std::stringstream ss;
            ss << "test_batchread_x_write3D<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
            std::cout << ss.str() << "> failed!! RESULT ERROR: Some results do not match baseline." << std::endl;
        } else {
            std::stringstream ss;
            ss << "test_batchread_x_write3D<" << cvTypeToString<CV_TYPE_I>() << ", " << cvTypeToString<CV_TYPE_O>();
            std::cout << ss.str() << "> failed!! EXCEPTION: " << error_s.str() << std::endl;
        }
    }

    return passed;
}

template <int CV_TYPE_I, int CV_TYPE_O, size_t... Is>
bool launch_test_batchread_x_write3D_only_HorizontalFusion(const size_t NUM_ELEMS_X, const size_t NUM_ELEMS_Y, std::index_sequence<Is...> seq, cv::cuda::Stream cv_stream, bool enabled) {
    bool passed = true;

    int dummy[] = { (passed &= test_batchread_x_write3D_only_HorizontalFusion<CV_TYPE_I, CV_TYPE_O, batchValues[Is]>(NUM_ELEMS_X, NUM_ELEMS_Y, cv_stream, enabled), 0)... };
    (void)dummy;

    return passed;
}

int launch() {
    return 0;
}